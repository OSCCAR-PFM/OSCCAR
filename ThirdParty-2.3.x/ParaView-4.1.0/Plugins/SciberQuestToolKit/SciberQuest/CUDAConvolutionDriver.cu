/*
   ____    _ __           ____               __    ____
  / __/___(_) /  ___ ____/ __ \__ _____ ___ / /_  /  _/__  ____
 _\ \/ __/ / _ \/ -_) __/ /_/ / // / -_|_-</ __/ _/ // _ \/ __/
/___/\__/_/_.__/\__/_/  \___\_\_,_/\__/___/\__/ /___/_//_/\__(_)

Copyright 2012 SciberQuest Inc.
*/

//#define CUDAConvolutionDriverDEBUG

#include "CUDAConvolutionDriver.h"

#include "SQVTKTemplateMacroWarningSupression.h"
#include "CartesianExtent.h"
#include "MemOrder.hxx"
#include "SQMacros.h"
#include "postream.h"

#if defined SQTK_CUDA
  #include "CUDAGlobalMemoryManager.hxx"
  #include "CUDAConstMemoryManager.hxx"
  #include "CUDANumerics.hxx"
  #include "CUDAMacros.h"
  #include <hip/hip_runtime.h>
  #include <hip/hip_runtime.h>
#endif

#include "vtkDataArray.h"

#include <iostream>
#include <vector>

//-----------------------------------------------------------------------------
CUDAConvolutionDriver::CUDAConvolutionDriver()
        :
     NDevices(0),
     DeviceId(0),
     MaxThreads(1),
     MaxBlocks(1),
     NThreads(1),
     NBlocks(1),
     KernelMemoryType(CUDA_MEM_TYPE_GLOBAL),
     InputMemoryType(CUDA_MEM_TYPE_GLOBAL),
     WarpSize(32),
     WarpsPerBlock(1),
     MaxWarpsPerBlock(1)
{
  int nDevs=0;
  #if defined SQTK_CUDA
  hipGetDeviceCount(&nDevs);
  #endif
  this->NDevices=nDevs;
}

//-----------------------------------------------------------------------------
int CUDAConvolutionDriver::SetDeviceId(int deviceId)
{
  #ifdef CUDAConvolutionDriverDEBUG
  pCerr()
    << "===============CUDAConvolutionDriver::SetDeviceId" << std::endl
    << deviceId << std::endl;
  #endif
  #if defined SQTK_CUDA
  if (deviceId>=this->NDevices)
    {
    sqErrorMacro(
      pCerr(),
      << "Attempt to select invalid device "
      << deviceId << " of " << this->NDevices);
    }

  this->DeviceId=deviceId;
  hipError_t ierr=hipSetDevice(deviceId);
  if (ierr)
    {
    CUDAErrorMacro(pCerr(),ierr,"Failed to select device " << deviceId);
    return -1;
    }

  hipDeviceProp_t props;
  hipGetDeviceProperties(&props, deviceId);

  this->MaxThreads=props.maxThreadsPerBlock;
  this->NThreads=props.maxThreadsPerBlock/2;

  this->MaxBlocks=props.maxGridSize[0];
  this->SetNumberOfBlocks(1024);

  this->BlockGridMax[0]=props.maxGridSize[0];
  this->BlockGridMax[1]=props.maxGridSize[1];
  this->BlockGridMax[2]=props.maxGridSize[2];
  this->WarpSize=props.warpSize;
  this->MaxWarpsPerBlock=props.maxThreadsPerBlock/props.warpSize;
  #endif

  return 0;
}

//-----------------------------------------------------------------------------
int CUDAConvolutionDriver::Convolution(
    CartesianExtent &extV,
    CartesianExtent &extW,
    CartesianExtent &extK,
    int nGhost,
    int mode,
    vtkDataArray *V,
    vtkDataArray *W,
    float *K)
{
  // TODO - make sure nothing is leaked if an error occurs!

  #ifdef CUDAConvolutionDriverDEBUG
  pCerr()
    << "===============CUDAConvolutionDriver::Convolution" << std::endl;
  #endif

  #if defined SQTK_CUDA
  int nV[3];
  extV.Size(nV);
  unsigned long vnijk=extV.Size();

  int nW[3];
  extW.Size(nW);
  unsigned long wnijk=extW.Size();

  int nK[3];
  extK.Size(nK);
  unsigned long knijk=extK.Size();

  CUDAMemoryManager<float> *devK;
  if ( (this->KernelMemoryType==CUDA_MEM_TYPE_CONST)
    && (knijk*sizeof(float)<65536) )
    {
    #ifdef CUDAConvolutionDriverDEBUG
    pCerr() << "Using constant memory for kernel" << std::endl;
    #endif
    devK=CUDAConstMemoryManager<float>::New("gK",K,knijk);
    }
  else
  if (this->KernelMemoryType==CUDA_MEM_TYPE_TEX)
    {
    // TODO
    sqErrorMacro(std::cerr,"Kernel texture memory is not implemented!");
    return -1;
    }
  else
    {
    #ifdef CUDAConvolutionDriverDEBUG
    pCerr() << "Using global memory for kernel" << std::endl;
    #endif
    devK=CUDAGlobalMemoryManager<float>::New(K,knijk);
    }

  unsigned long nComp=W->GetNumberOfComponents();

  dim3 blockGrid(1,1,1);
  dim3 threadGrid(1,1,1);
  unsigned long nBlocks;
  int ierr=PartitionBlocks(
        wnijk,
        this->WarpsPerBlock,
        this->WarpSize,
        this->BlockGridMax,
        blockGrid,
        nBlocks,
        threadGrid);
  if (ierr)
    {
    sqErrorMacro(pCerr(),"Failed to decompose domain for the GPU");
    return -1;
    }


  int fastDim=0;
  int slowDim=1;
  switch (mode)
    {
    case CartesianExtent::DIM_MODE_2D_XY:
      fastDim=0;
      slowDim=1;
      break;
    case CartesianExtent::DIM_MODE_2D_XZ:
      fastDim=0;
      slowDim=2;
      break;
    case CartesianExtent::DIM_MODE_2D_YZ:
      fastDim=1;
      slowDim=2;
      break;
    case CartesianExtent::DIM_MODE_3D:
      fastDim=0;
      slowDim=1;
      break;
    default:
      sqErrorMacro(std::cerr,"Bad dim mode.");
      return -1;
    }

  #ifdef CUDAConvolutionDriverDEBUG
  pCerr() << "wnijk=" << wnijk << std::endl;
  pCerr() << "WarpsPerBlock=" << this->WarpsPerBlock << std::endl;
  pCerr() << "WarpSize=" << this->WarpSize << std::endl;
  pCerr() << "blockGridMaxMax=(" << this->BlockGridMax[0] << ", " << this->BlockGridMax[1] << ", " << this->BlockGridMax[2] << ")" << std::endl;
  pCerr() << "blockGrid=(" << blockGrid.x << ", " << blockGrid.y << ", " << blockGrid.z << ")" << std::endl;
  pCerr() << "nBlocks=" << nBlocks << std::endl;
  pCerr() << "threadGrid=(" << threadGrid.x << ", " << threadGrid.y << ", " << threadGrid.z << ")" << std::endl;
  pCerr() << "fastDim=" << fastDim << std::endl;
  pCerr() << "slowDim=" << slowDim << std::endl;
  pCerr() << "extV=" << extV << std::endl;
  pCerr() << "nV=(" << nV[fastDim] <<  ", " << nV[slowDim] << ")" << std::endl;
  pCerr() << "extW=" << extW << std::endl;
  pCerr() << "nW=(" << nW[fastDim] <<  ", " << nW[slowDim] << ")" << std::endl;
  #endif

  switch (V->GetDataType())
    {
    // TODO -- replace with vtkTemplateMacro
    case VTK_FLOAT:
      {
      //int worldRank;
      //MPI_Comm_rank(MPI_COMM_WORLD,&worldRank);
      hipError_t uerr;
      // allocate device memory for vector components
      std::vector<float*> sV(nComp,0);
      std::vector<CUDAGlobalMemoryManager<float>*> devV(nComp,0);
      std::vector<CUDAGlobalMemoryManager<float>*> devW(nComp,0);
      std::vector<float*> sW(nComp,0);
      for (int q=0; q<nComp; ++q)
        {
        // input arrays
        hipHostAlloc(
              &sV[q],
              vnijk*sizeof(float),
              hipHostMallocDefault);
        if (this->InputMemoryType==CUDA_MEM_TYPE_TEX)
          {
          // use texture memory for input
          if ((mode==CartesianExtent::DIM_MODE_2D_XY)
            ||(mode==CartesianExtent::DIM_MODE_2D_XZ)
            ||(mode==CartesianExtent::DIM_MODE_2D_YZ))
            {
            devV[q]
              = CUDAGlobalMemoryManager<float>::New(
                  nV[fastDim],
                  nV[slowDim]);
            }
          else
            {
            // TODO
            sqErrorMacro(std::cerr,"3D Texture kernel is not implemented!");
            return -1;
            }
          }
        else
          {
          // use global memory for input
          devV[q]=CUDAGlobalMemoryManager<float>::New(vnijk);
          }

        // output array
        devW[q]=CUDAGlobalMemoryManager<float>::New(wnijk);
        hipHostAlloc(
            &sW[q],
            wnijk*sizeof(float),
            hipHostMallocDefault);

        /*
        // fill device data with falt index for lookup validation
        for (int i=0; i<vnijk; ++i)
          {
          sV[q][i]=i;
          }
        */
        }

      // convert vtk vectors/tensors into scalar component arrays
      float *hV=(float*)V->GetVoidPointer(0);
      Split<float>(vnijk,hV,sV);

      // copy the input arrays to the device
      for (int q=0; q<nComp; ++q)
        {
        // TODO-could this be streamed to overlap com w/ comp?
        devV[q]->Push(sV[q]);

        // execute the kernel
        if (this->InputMemoryType==CUDA_MEM_TYPE_TEX)
          {
          #ifdef CUDAConvolutionDriverDEBUG
          pCerr() << "Using texture memory for input" << std::endl;
          #endif
            if ((mode==CartesianExtent::DIM_MODE_2D_XY)
              ||(mode==CartesianExtent::DIM_MODE_2D_XZ)
              ||(mode==CartesianExtent::DIM_MODE_2D_YZ))
              {
              // setup texture
              ierr=devV[q]->BindToTexture(&gV2);
              CUDAScalarConvolution2D<<<blockGrid,threadGrid>>>(
                    //worldRank,
                    nV[fastDim],
                    nW[fastDim],
                    wnijk,
                    nK[fastDim],
                    nGhost,
                    devW[q]->GetDevicePointer(),
                    devK->GetDevicePointer());
              }
            else
              {
              // TODO
              sqErrorMacro(std::cerr,"3D Texture kernel is not implemented!");
              return -1;
              }
            uerr=hipGetLastError();
            if (uerr)
              {
              CUDAErrorMacro(pCerr(),uerr,"Error invoking 2D kernel.");
              return -1;
              }
            // release texture
            hipDeviceSynchronize();
            //hipUnbindTexture(gV2);
          }
        else
          {
          #ifdef CUDAConvolutionDriverDEBUG
          pCerr() << "Using global memory for input" << std::endl;
          #endif
          if ((mode==CartesianExtent::DIM_MODE_2D_XY)
            ||(mode==CartesianExtent::DIM_MODE_2D_XZ)
            ||(mode==CartesianExtent::DIM_MODE_2D_YZ))
            {
            CUDAScalarConvolution2D<<<blockGrid,threadGrid>>>(
                  //worldRank,
                  nV[fastDim],
                  nW[fastDim],
                  wnijk,
                  nK[fastDim],
                  nGhost,
                  devV[q]->GetDevicePointer(),
                  devW[q]->GetDevicePointer(),
                  devK->GetDevicePointer());
            }
          else
            {
            CUDAScalarConvolution3D<<<blockGrid,threadGrid>>>(
                  nV[fastDim],
                  nV[fastDim]*nV[slowDim],
                  nW[fastDim],
                  nW[fastDim]*nW[slowDim],
                  wnijk,
                  nK[fastDim],
                  nK[fastDim]*nK[slowDim],
                  nGhost,
                  devV[q]->GetDevicePointer(),
                  devW[q]->GetDevicePointer(),
                  devK->GetDevicePointer());
            }
          uerr=hipGetLastError();
          if (uerr)
            {
            CUDAErrorMacro(pCerr(),uerr,"Error invoking global memory kernel.");
            return -1;
            }
          }
        // retreive the output/result
        devW[q]->Pull(sW[q]);
        }

      // put results in vtk order
      float *hW=(float*)W->GetVoidPointer(0);
      Interleave(wnijk,sW,hW);

      // clean up
      for (int q=0; q<nComp; ++q)
        {
        delete devW[q];
        hipHostFree(sW[q]);
        delete devV[q];
        hipHostFree(sV[q]);
        }
      hipUnbindTexture(gV2);
      }
      break;
    default:
      // TODO
      sqErrorMacro(std::cerr,"Not currently using vtkTemplateMacro");
      return -1;
    }

  delete devK;
  // TODO if kernel is in texture mem, unbind
  #endif

  return 0;
}

/*
//-----------------------------------------------------------------------------
void CUDAConvolutionDriver::Convolution(
    CartesianExtent &extV,
    CartesianExtent &extW,
    CartesianExtent &extK,
    int ghostV,
    int mode,
    vtkDataArray *V,
    vtkDataArray *W,
    float *K)
{
  pCerr()
    << "===============CUDAConvolutionDriver::Convolution" << std::endl
    << "NBlocks=" << this->NBlocks << std::endl
    << "NThreads=" << this->NThreads << std::endl;

  CUDAFlatIndex  idxV(extV,mode);
  CUDATupleIndex tupW(extW,ghostV,mode);
  CUDAFlatIndex  idxK(extK,mode);

  int compW=W->GetNumberOfComponents();
  unsigned long sizeW=W->GetNumberOfTuples();

  CUDAMemoryManager<int> *devExtK
    = CUDAMemoryManager<int>::New(extK.GetData(),6);
  devExtK->Push();

  CUDAMemoryManager<float> *devK
    = CUDAMemoryManager<float>::New(K,extK.Size());
  devK->Push();

  switch (V->GetDataType())
    {
    case VTK_FLOAT:
      {
      CUDAMemoryManager<float> *devV
        = CUDAMemoryManager<float>::New(V);
      devV->Push();

      CUDAMemoryManager<float> *devW
        = CUDAMemoryManager<float>::New(W);
      //devW->Zero(); bug in cuda!
      //devW->Push();

      std::cerr << "Calling" << std::endl;
      ::CUDAConvolution<float><<<this->NBlocks,this->NThreads>>>(
          idxV.GetDevicePointer(),
          devV->GetDevicePointer(),
          tupW.GetDevicePointer(),
          sizeW,
          compW,
          devW->GetDevicePointer(),
          devExtK->GetDevicePointer(),
          idxK.GetDevicePointer(),
          devK->GetDevicePointer());
      hipError_t ierr=hipGetLastError();
      if (ierr)
      {
        CUDAErrorMacro(pCerr(),ierr,"Kernel fialed to run.");
      }

      devW->Pull();

      delete devW;
      delete devV;
      }
      break;
    };

  delete devExtK;
  delete devK;
  std::cerr << "Finished." << std::endl;
}

//-----------------------------------------------------------------------------
void CUDAConvolutionDriver::Convolution3D(
    CartesianExtent &extV,
    CartesianExtent &extW,
    CartesianExtent &extK,
    int nGhost,
    int mode,
    vtkDataArray *V,
    vtkDataArray *W,
    float *K)
{
  // block decomp
  int wSize[3];
  extW.Size(wSize);
  dim3 blockDecomp(wSize[0],wSize[1],wSize[2]);

  // thread decomp
  int nCompV=V->GetNumberOfComponents();

  // translation between output and input index tuple
  int nGhostV[3]={0};
  CartesianExtent::Shift(nGhostV,nGhost,mode);
  CUDAMemoryManager<int> *devNGhostV
    = CUDAMemoryManager<int>::New(nGhostV,3);
  devNGhostV->Push();

  //
  CUDAFlatIndex idxV(extV,mode);
  CUDAFlatIndex idxW(extW,mode);
  CUDAFlatIndex idxK(extK,mode);

  CUDAMemoryManager<int> *devExtK
    = CUDAMemoryManager<int>::New(extK.GetData(),6);
  devExtK->Push();

  CUDAMemoryManager<float> *devK
    = CUDAMemoryManager<float>::New(K,extK.Size());
  devK->Push();

  pCerr()
    << "===============CUDAConvolutionDriver::Convolution3D" << std::endl
    << "NBlocks=" << wSize[0] << ", " << wSize[1] << ", " << wSize[2] << std::endl
    << "NThreads=" << nCompV << std::endl;

  switch (V->GetDataType())
    {
    case VTK_FLOAT:
      {
      CUDAMemoryManager<float> *devV
        = CUDAMemoryManager<float>::New(V);
      devV->Push();

      CUDAMemoryManager<float> *devW
        = CUDAMemoryManager<float>::New(W);
      //devW->Zero(); bug in cuda! not all bytes are zerod
      //devW->Push();

      std::cerr << "Calling" << std::endl;
      ::CUDAConvolution<float><<<blockDecomp,nCompV>>>(
          idxV.GetDevicePointer(),
          nCompV,
          devNGhostV->GetDevicePointer(),
          devV->GetDevicePointer(),
          idxW.GetDevicePointer(),
          devW->GetDevicePointer(),
          devExtK->GetDevicePointer(),
          idxK.GetDevicePointer(),
          devK->GetDevicePointer());
      hipDeviceSynchronize();
      hipError_t ierr=hipGetLastError();
      if (ierr!=hipSuccess)
      {
        CUDAErrorMacro(pCerr(),ierr,"Kernel fialed.");
      }

      devW->Pull();

      delete devW;
      delete devV;
      }
      break;
    };

    delete devK;
    delete devExtK;
    std::cerr << "Finished." << std::endl;
}
*/
