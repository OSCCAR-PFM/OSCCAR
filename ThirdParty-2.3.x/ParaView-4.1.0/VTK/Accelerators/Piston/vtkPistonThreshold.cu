#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include "vtkgl.h"
#include <piston/threshold_geometry.h>
#include <piston/vtk_image3d.h>
#include <iostream>
#include "vtkPistonDataObject.h"
#include "vtkPistonDataWrangling.h"
#include "vtkPistonReference.h"

using namespace std;
using namespace piston;

namespace vtkpiston {

// execution method found in vtkPistonThreshold.cu
void ExecutePistonThreshold(vtkPistonDataObject *inData,
                            float minvalue, float maxvalue,
                            vtkPistonDataObject *outData)
{
  vtkPistonReference *ti = inData->GetReference();
  if (ti->type != VTK_IMAGE_DATA || ti->data == NULL)
    {
    //cerr << "NVM" << endl;
    //type mismatch, don't bother trying
    return;
    }
  vtk_image3d<SPACE>*gpuData = (vtk_image3d<SPACE>*)ti->data;
  threshold_geometry<vtk_image3d<SPACE> > pistonFunctor(*gpuData, minvalue, maxvalue);

  // execute the PISTON filter
  pistonFunctor();

  vtkPistonReference *to = outData->GetReference();
  DeleteData(to);

  to->type = VTK_POLY_DATA;
  vtk_polydata *newD = new vtk_polydata;
  to->data = (void*)newD;
  //geometry
  newD->nPoints = pistonFunctor.vertices_indices.size();
  newD->vertsPer = 4; //this piston filter produces quads
  newD->points = new thrust::device_vector<float>(newD->nPoints*3);
  thrust::device_vector<float3> *tmp = (thrust::device_vector<float3> *)newD->points;
  thrust::copy(thrust::make_transform_iterator(pistonFunctor.vertices_begin(), float4tofloat3()),
               thrust::make_transform_iterator(pistonFunctor.vertices_end(), float4tofloat3()),
               tmp->begin());
  //attributes
  newD->scalars = new thrust::device_vector<float>(newD->nPoints);
  thrust::copy(pistonFunctor.scalars_begin(), pistonFunctor.scalars_end(), newD->scalars->begin());
  outData->SetScalarsArrayName(inData->GetScalarsArrayName());
  newD->normals = new thrust::device_vector<float>(newD->nPoints*3);
  //TODO: FIX UP THIS CAST
  thrust::device_vector<float3>* casted = reinterpret_cast<thrust::device_vector<float3>*>(newD->normals);
  thrust::copy(pistonFunctor.normals_begin(), pistonFunctor.normals_end(), casted->begin());
}

} //namespace
