#include "hip/hip_runtime.h"
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include "vtkgl.h"

#include <piston/marching_cube.h>
#include <piston/vtk_image3d.h>
#include <piston/vtk_plane_field.h>

#include "vtkPistonDataObject.h"
#include "vtkPistonDataWrangling.h"
#include "vtkPistonReference.h"
#include <iostream>

using namespace std;
using namespace piston;

namespace vtkpiston {

void ExecutePistonSlice(vtkPistonDataObject *inData,
                        float* origin, float*normal, float offset,
                        vtkPistonDataObject *outData)
{
  vtkPistonReference *ti = inData->GetReference();
  if (ti->type != VTK_IMAGE_DATA || ti->data == NULL)
    {
    // type mismatch, don't bother trying
    return;
    }
  vtk_image3d<SPACE>*gpuData =
      (vtk_image3d<SPACE>*)ti->data;

  float dataOrigin[3];
  dataOrigin[0] = static_cast<float>(gpuData->origin[0]);
  dataOrigin[1] = static_cast<float>(gpuData->origin[1]);
  dataOrigin[2] = static_cast<float>(gpuData->origin[2]);

  float dataSpacing[3];
  dataSpacing[0] = static_cast<float>(gpuData->spacing[0]);
  dataSpacing[1] = static_cast<float>(gpuData->spacing[1]);
  dataSpacing[2] = static_cast<float>(gpuData->spacing[2]);

  int dims[3];
  dims[0] = gpuData->dim0;
  dims[1] = gpuData->dim1;
  dims[2] = gpuData->dim2;

  vtk_plane_field<int, float, SPACE > *plane =
    new vtk_plane_field<int, float, SPACE >(
      dataOrigin, normal, dims, dataSpacing, gpuData->extents);

  marching_cube<vtk_plane_field<int, float, SPACE>,
    vtk_image3d<SPACE> > pistonFunctor(*plane, *gpuData, offset);

  // Execute the piston filter
  pistonFunctor();

  vtkPistonReference *to = outData->GetReference();
  DeleteData(to);

  to->type = VTK_POLY_DATA;
  vtk_polydata *newD = new vtk_polydata;
  to->data = (void*)newD;
  //geometry
  newD->nPoints = pistonFunctor.vertices.size();
  newD->vertsPer = 3; //this piston filter produces triangles
  newD->points = new thrust::device_vector<float>(newD->nPoints*3);
  thrust::device_vector<float3> *tmp =
    (thrust::device_vector<float3> *)newD->points;
  thrust::copy(thrust::make_transform_iterator(pistonFunctor.vertices_begin(),
                                               float4tofloat3()),
               thrust::make_transform_iterator(pistonFunctor.vertices_end(),
                                               float4tofloat3()),
               tmp->begin());
  //attributes
  newD->scalars = new thrust::device_vector<float>(newD->nPoints);
  thrust::copy(pistonFunctor.scalars_begin(), pistonFunctor.scalars_end(),
               newD->scalars->begin());
  outData->SetScalarsArrayName(inData->GetScalarsArrayName());
  newD->normals = new thrust::device_vector<float>(newD->nPoints*3);
  //TODO: FIX UP THIS CAST
  thrust::device_vector<float3>* casted =
    reinterpret_cast<thrust::device_vector<float3>*>(newD->normals);
  thrust::copy(pistonFunctor.normals_begin(), pistonFunctor.normals_end(),
               casted->begin());
}

} //namespace
