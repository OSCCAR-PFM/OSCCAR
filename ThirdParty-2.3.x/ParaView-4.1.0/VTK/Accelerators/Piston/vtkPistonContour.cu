#include "hip/hip_runtime.h"
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include "vtkgl.h"
#include <piston/marching_cube.h>
#include <piston/vtk_image3d.h>
#include "vtkPistonDataObject.h"
#include "vtkPistonReference.h"
#include "vtkPistonDataWrangling.h"

#include <iostream>

namespace vtkpiston {

using namespace std;
using namespace piston;

// execution method found in vtkPistonContour.cu
void ExecutePistonContour(vtkPistonDataObject *inData,
                          float isovalue,
                          vtkPistonDataObject *outData)
{
  vtkPistonReference *ti = inData->GetReference();
  if (ti->type != VTK_IMAGE_DATA || ti->data == NULL)
    {
    //type mismatch, don't bother trying
    return;
    }
  vtk_image3d<SPACE>*gpuData = (vtk_image3d<SPACE>*)ti->data;
  marching_cube<vtk_image3d<SPACE>, vtk_image3d<SPACE> >
    pistonFunctor(*gpuData, *gpuData, isovalue);

  // Execute the piston filter
  pistonFunctor();

  vtkPistonReference *to = outData->GetReference();
  DeleteData(to);

  to->type = VTK_POLY_DATA;
  vtk_polydata *newD = new vtk_polydata;
  to->data = (void*)newD;
  //geometry
  newD->nPoints = pistonFunctor.vertices.size();
  newD->vertsPer = 3; //this piston filter produces triangles
  newD->points = new thrust::device_vector<float>(newD->nPoints*3);
  thrust::device_vector<float3> *tmp = (thrust::device_vector<float3> *)newD->points;
  thrust::copy(thrust::make_transform_iterator(pistonFunctor.vertices_begin(), float4tofloat3()),
               thrust::make_transform_iterator(pistonFunctor.vertices_end(), float4tofloat3()),
               tmp->begin());
  //attributes
  newD->scalars = new thrust::device_vector<float>(newD->nPoints);
  outData->SetScalarsArrayName(inData->GetScalarsArrayName());
  thrust::copy(pistonFunctor.scalars_begin(), pistonFunctor.scalars_end(), newD->scalars->begin());
  newD->normals = new thrust::device_vector<float>(newD->nPoints*3);
  //TODO: FIX UP THIS CAST
  thrust::device_vector<float3>* casted = reinterpret_cast<thrust::device_vector<float3>*>(newD->normals);
  thrust::copy(pistonFunctor.normals_begin(), pistonFunctor.normals_end(), casted->begin());
}

} //namespace
